// 🚍 Otobüs Rotalama Optimizasyonu - C++ ve CUDA Hazırlık
// Amaç: Popülasyon üzerinden genetik algoritma ve paralel fitness hesaplama

#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#define NUM_STOPS 10
#define POP_SIZE 30
#define MAX_GENERATIONS 100

struct Individual {
    int bus_id;
    int route[NUM_STOPS];
    int departure_time;
};

int distance_matrix[NUM_STOPS][NUM_STOPS];
int passenger_demand[NUM_STOPS];
int bus_capacities[5] = {40, 30, 50, 45, 35};

float fitness_scores[POP_SIZE];

void generate_distance_matrix() {
    for (int i = 0; i < NUM_STOPS; ++i) {
        for (int j = 0; j < NUM_STOPS; ++j) {
            distance_matrix[i][j] = rand() % 10 + 1;
        }
    }
}

void generate_passenger_demand() {
    for (int i = 0; i < NUM_STOPS; ++i) {
        passenger_demand[i] = rand() % 11 + 5;
    }
}

Individual create_individual() {
    Individual ind;
    ind.bus_id = rand() % 5;
    for (int i = 0; i < NUM_STOPS; ++i) ind.route[i] = i;
    std::random_shuffle(ind.route, ind.route + NUM_STOPS);
    ind.departure_time = rand() % 5 + 6;
    return ind;
}

void create_population(std::vector<Individual>& population) {
    for (int i = 0; i < POP_SIZE; ++i) {
        population.push_back(create_individual());
    }
}

__device__ float calculate_fitness_gpu(const Individual& ind, const int* dist_matrix, const int* demand, const int* capacities) {
    int total_distance = 0;
    int total_passengers = 0;
    for (int i = 0; i < NUM_STOPS - 1; ++i) {
        int from = ind.route[i];
        int to = ind.route[i + 1];
        total_distance += dist_matrix[from * NUM_STOPS + to];
    }
    for (int i = 0; i < NUM_STOPS; ++i) {
        total_passengers += demand[ind.route[i]];
    }
    int capacity = capacities[ind.bus_id];
    int penalty = (total_passengers > capacity) ? (total_passengers - capacity) * 10 : 0;
    return 1.0f / (total_distance + 1 + penalty);
}

__global__ void calculate_fitness_kernel(Individual* population, float* fitness, const int* dist_matrix, const int* demand, const int* capacities) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < POP_SIZE) {
        fitness[idx] = calculate_fitness_gpu(population[idx], dist_matrix, demand, capacities);
    }
}

int main() {
    srand(time(0));
    std::vector<Individual> population;
    generate_distance_matrix();
    generate_passenger_demand();
    create_population(population);

    // CUDA için bellek tahsisi ve veri kopyalama
    Individual* d_population;
    float* d_fitness;
    int* d_dist_matrix;
    int* d_demand;
    int* d_capacities;

    hipMalloc(&d_population, sizeof(Individual) * POP_SIZE);
    hipMalloc(&d_fitness, sizeof(float) * POP_SIZE);
    hipMalloc(&d_dist_matrix, sizeof(int) * NUM_STOPS * NUM_STOPS);
    hipMalloc(&d_demand, sizeof(int) * NUM_STOPS);
    hipMalloc(&d_capacities, sizeof(int) * 5);

    hipMemcpy(d_population, population.data(), sizeof(Individual) * POP_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_dist_matrix, distance_matrix, sizeof(int) * NUM_STOPS * NUM_STOPS, hipMemcpyHostToDevice);
    hipMemcpy(d_demand, passenger_demand, sizeof(int) * NUM_STOPS, hipMemcpyHostToDevice);
    hipMemcpy(d_capacities, bus_capacities, sizeof(int) * 5, hipMemcpyHostToDevice);

    // CUDA kernel çağır
    calculate_fitness_kernel<<<(POP_SIZE + 255)/256, 256>>>(d_population, d_fitness, d_dist_matrix, d_demand, d_capacities);
    hipDeviceSynchronize();

    // Sonuçları host'a kopyala
    hipMemcpy(fitness_scores, d_fitness, sizeof(float) * POP_SIZE, hipMemcpyDeviceToHost);

    std::cout << "Ilk bireyin fitness puani (GPU): " << fitness_scores[0] << std::endl;

    // Bellek temizle
    hipFree(d_population);
    hipFree(d_fitness);
    hipFree(d_dist_matrix);
    hipFree(d_demand);
    hipFree(d_capacities);

    return 0;
}